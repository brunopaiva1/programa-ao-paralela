/* 17. [Pacheco and Malensek, 2022] Implemente uma ordenação bitônica na qual cada thread
é responsável por dois blocos de elementos. Se o array tiver n elementos e houver
blk_ct blocos de threads e th_per_blk threads por bloco, considere que o número total
de threads é uma potência de dois e que n é divisível pelo número de threads. Assim
chunk_sz = n
blk_ct ×th_per_blk (2)
é um inteiro.
Cada thread é responsável por uma sublista contígua de chunk_sz elementos, e cada
thread inicialmente classificará sua sublista em ordem crescente. Então, se as threads t
e u forem pareadas para uma divisão e mesclagem, t < u, e t e u estiverem trabalhando
em uma sequência crescente, elas mesclarão suas sublistas em uma sequência cres-
cente, com t mantendo a metade inferior e u mantendo a metade superior. Se estiverem
trabalhando em uma sequência decrescente, t manterá a metade superior e u manterá a
metade inferior. Portanto, após cada divisão e mesclagem, cada thread sempre terá uma
sublista crescente.
Primeiro implemente a ordenação bitônica usando um único bloco de threads. Em se-
guida, modifique o programa para que ele possa lidar com um número arbitrário de blocos
de threads */

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define THREADS_PER_BLOCK 256

int check_order(int *array, int n) {
    for (int i = 0; i < n - 1; i++) {
        if (array[i] > array[i + 1]) {
            return 0;
        }
    }
    return 1;
}

int pot_2(int n) {
    int pot = 1;
    while (pot < n) {
        pot *= 2;
    }
    return pot;
}

int *generate_random_array(int n, int size) {
    int *array = (int *)malloc(size * sizeof(int));
    if (array == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < n; i++) {
        array[i] = rand() % (INT_MAX/10000000);
    }

    for(int i = n; i < size; i++) {
        array[i] = INT_MAX;
    }
    return array;
}

void imprimi_array(int *array, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d ", array[i]);
    }
    printf("\n");
}


